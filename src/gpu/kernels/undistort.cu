#include "hip/hip_runtime.h"
#include "undistort.h"
#include "cudamath.h"
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime_api.h>

//!
//! \brief Apply distortion correction as in
//! OpenCV undistort
//! \param x original x coordinates
//! \param y original y coordinates
//! \param length number of coordinates (i.e.
//! length of \c x and \c y)
//! \param k1 see OpenCV undistort
//! \param k2 see OpenCV undistort
//! \param k3 see OpenCV undistort
//! \param fx focal length (x) of camera
//! matrix
//! \param fy focal length (y) of camera
//! matrix
//! \param cx principal point (x) of camera
//! matrix
//! \param cy principal point (y) of camera
//! matrix
//! \param x_u distortion-corrected x
//! coordinates
//! \param y_u distortion-corrected y
//! coordinates
//!
__global__ void undistort(const float * x, const float * y,
                          size_t length,
                          const float k1, const float k2, const float k3,
                          const float fx, const float fy,
                          const float cx, const float cy,
                          float * x_u, float * y_u)
{
    float r = 0, tmp = 0;
    for (size_t i = 0; i < length; i++) {
        r = powf(x[i], 2) + powf(y[i], 2);
        // common coefficient for multiplication
        tmp = 1 + k1 * powf(r, 2) + k2 * powf(r, 4) + k3 * powf(r, 6);

        x_u[i] = (x[i] - cx) / fx;
        x_u[i] *= tmp;
        y_u[i] = (y[i] - cy) / fy;
        y_u[i] *= tmp;
    }
}

void cuda_undistort(const float * x, const float * y,
                    const size_t cols, const size_t rows,
                    const float * camera_matrix,
                    const float * distortion_coeffs,
                    float * x_u, float * y_u,
                    hipStream_t stream)
{
    // TODO blocks
    dim3 blocks(16, 16);
    // TODO grid
    dim3 grid(DivUp(cols, blocks.x), DivUp(rows, blocks.y));
    float k1 = distortion_coeffs[0],
          k2 = distortion_coeffs[1],
          k3 = distortion_coeffs[2];
    float fx = camera_matrix[0],
          fy = camera_matrix[1],
          cx = camera_matrix[2],
          cy = camera_matrix[3];
    undistort<<<grid, blocks, 0, stream>>>(x, y,
                                           rows * cols,
                                           k1, k2, k3,
                                           fx, fy,
                                           cx, cy,
                                           x_u, y_u);
    getLastCudaError("Undistort kernel launch failed");
    // TODO add resample!
}
