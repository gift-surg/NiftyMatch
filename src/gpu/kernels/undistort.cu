#include "hip/hip_runtime.h"
#include "undistort.h"
#include "cudamath.h"
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime_api.h>

//!
//! \brief Apply distortion correction as in
//! OpenCV undistort
//! \param x original x coordinates
//! \param y original y coordinates
//! \param cols
//! \param rows
//! \param distortion coeffs k1, k2, k3 as in
//! OpenCV undistort
//! \param camera_matrix fx, fy, cx, cy as in
//! OpenCV undistort
//! \param u distortion-corrected x
//! coordinates (no boundary check performed!)
//! \param v distortion-corrected y
//! coordinates (no boundary check performed!)
//!

//!
//! \brief Compute distortion correction map
//! from undistorted image coordinates \c u and
//! \c v to original image coordinates \c x and
//! \c y, using same process as OpenCV undistort)
//! \param x original x coordinates (used as
//! initial values for \c u)
//! \param y original y coordinates (used as
//! initial values for \c v)
//! \param cols not used for border checks, so
//! resulting \c u values might be out of
//! bounds!
//! \param rows not used for border checks, so
//! resulting \c v values might be out of
//! bounds!
//! \param camera_matrix fx, fy, cx, and cy as
//! in OpenCV undistort
//! \param distortion_coeffs k1, k2, and k3 as
//! in OpenCV undistort
//! \param u for each new position \c u_i, use
//! \c u[u_i] from original image
//! \param v for each new potision \c v_i, use
//! \c v[v_i] from original image
//!
__global__ void undistort(const float * x, const float * y,
                          const size_t cols, const size_t rows,
                          const float * distortion_coeffs,
                          const float * camera_matrix,
                          float * u, float * v)
{
    const size_t i = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const size_t j = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    const size_t pos = j * cols + i; // cols = width, rows = height

    const float k1 = distortion_coeffs[0],
                k2 = distortion_coeffs[1],
                k3 = distortion_coeffs[3];
    const float fx = camera_matrix[0],
                fy = camera_matrix[1],
                cx = camera_matrix[2],
                cy = camera_matrix[3];

    if (0 <= pos and pos < cols * rows) {

        u[pos] = x[pos];
        u[pos] -= cx;
        u[pos] /= fx;

        v[pos] = y[pos];
        v[pos] -= cy;
        v[pos] /= fy;

        // radial distortion correction, at this point:
        // u = x', and v = y'
        float r2 = powf(u[pos], 2) + powf(v[pos], 2);
        float kr_poly = 1 + k1 * r2 + k2 * powf(r2,2) + k3 * powf(r2,3);

        u[pos] /= kr_poly;
        u[pos] *= fx;
        u[pos] += cx;

        v[pos] /= kr_poly;
        v[pos] *= fy;
        v[pos] += cy;
    }
}

void cuda_undistort(const float * x, const float * y,
                    const size_t cols, const size_t rows,
                    const float * camera_matrix,
                    const float * distortion_coeffs,
                    float * u, float * v,
                    hipStream_t stream)
{
    dim3 blocks(16, 16);
    dim3 grid(DivUp(cols, blocks.x), DivUp(rows, blocks.y));
    undistort<<<grid, blocks, 0, stream>>>(x, y,
                                           cols, rows,
                                           distortion_coeffs,
                                           camera_matrix,
                                           u, v);
    getLastCudaError("Undistort kernel launch failed");
    // TODO add resample!
}
