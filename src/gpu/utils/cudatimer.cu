#include "cudatimer.h"

CudaTimer::CudaTimer(hipStream_t stream)
    :_stream(stream)
{
    hipEventCreate(&_start);
    hipEventCreate(&_stop);
}

void CudaTimer::start()
{
    hipEventRecord(_start, _stream);
}

float CudaTimer::stop()
{
    hipEventRecord(_stop, _stream);
    hipEventSynchronize(_stop);
    float et;
    hipEventElapsedTime(&et, _start, _stop);
    return et;
}
