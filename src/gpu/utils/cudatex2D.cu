#include "cudatex2D.h"
#include "hip/hip_runtime_api.h"

void CudaTex2D::set(hipArray *array, hipTextureReadMode read_mode)
{
    release();

    hipResourceDesc res_desc;
    memset(&res_desc, 0, sizeof(res_desc));
    res_desc.resType = hipResourceTypeArray;
    res_desc.res.array.array = array;

    hipTextureDesc tex_desc;
    memset(&tex_desc, 0, sizeof(tex_desc));
    tex_desc.addressMode[0] = hipAddressModeBorder;
    tex_desc.addressMode[1] = hipAddressModeBorder;
    tex_desc.filterMode = hipFilterModeLinear;
    tex_desc.readMode = read_mode;
    tex_desc.normalizedCoords = false;
    checkCudaErrors(hipCreateTextureObject(&_tex, &res_desc, &tex_desc, NULL));
}

CudaTex2D::CudaTex2D(hipArray * array)
{
    set(array);
}

void CudaTex2D::release()
{
    if (_tex) {
        hipDestroyTextureObject(_tex);
        _tex = 0;
    }
}

CudaTex2D::~CudaTex2D()
{
    hipDestroyTextureObject(_tex);
}
